#include "hip/hip_runtime.h"
#include <func.hpp>

#define CUDA_CHECK(val) { \
	if (val != hipSuccess) { \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
		exit(val); \
	} \
}

using namespace std;


int main(){
    clock_t g_start, g_end;
    graph_created=false;
    int size =1<<10;
    printf("%d\n",size);
    hipStream_t stream0;
    hipStream_t stream1;
    CUDA_CHECK(hipStreamCreate(&stream0));
    CUDA_CHECK(hipStreamCreate(&stream1));
    /*init host memory*/
    int* h_data0;
    int* h_data1;
    int* h_result;
    h_data0 = (int*)calloc(size*3, sizeof(int));
    h_data1 = &h_data0[size];
    h_result = &h_data0[size*2];

    srand(990720);
    for (int i=0;i<size;i++){
        h_data0[i]= rand()%10;
    }
    // Aprint(h_data0, size);
    // Aprint(h_data1, size);

    /*init device memory*/
    int* d_data0;
    int* d_data1;
    int* d_result;
    CUDA_CHECK(hipMalloc((void**)&d_data0, (size*3)*sizeof(int)));
    d_data1=&d_data0[size];
    d_result=&d_data0[size*2];

    /*memcpyH2D*/
    CUDA_CHECK(hipMemcpyAsync(d_data0, h_data0, size*3*sizeof(int), hipMemcpyHostToDevice, stream0));
    CUDA_CHECK(hipStreamBeginCapture(stream0, hipStreamCaptureModeGlobal));
    adding(d_data0, stream0, size);
    CUDA_CHECK(hipStreamEndCapture(stream0, &graph));
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    graph_created=true;
    //printf("graph init\n");
    g_start = clock();
    for (int i=0; i<100000; i++){
    CUDA_CHECK(hipGraphLaunch(graphExec, stream0));
    CUDA_CHECK(hipStreamSynchronize(stream0));
    }
    g_end = clock();
    printf("total %lf s\n", ((float)(g_end - g_start)/1000000));
    
    /*memcpyD2H*/
    CUDA_CHECK(hipMemcpyAsync(h_data0, d_data0, size*3*sizeof(int), hipMemcpyDeviceToHost, stream0));
    Aprint(h_data0, size);
    // Aprint(h_data1, size);
    // Aprint(h_result, size);

    /*free*/
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    CUDA_CHECK(hipFree(d_data0));
    CUDA_CHECK(hipStreamDestroy(stream0));
    CUDA_CHECK(hipStreamDestroy(stream1));
    free(h_data0);
    return 0;
} 

