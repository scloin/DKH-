#include "hip/hip_runtime.h"
#include <func.hpp>
std::vector<hipGraphNode_t> _node_list;
#define CUDA_CHECK(val) { \
	if (val != hipSuccess) { \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
		exit(val); \
	} \
}

using namespace std;


int main(){
    clock_t g_start, g_end;
    graph_created=false;
    int size =1<<10;
    printf("%d\n",size);
    hipStream_t stream0;
    hipStream_t stream1;
    CUDA_CHECK(hipStreamCreate(&stream0));
    CUDA_CHECK(hipStreamCreate(&stream1));
    /*init host memory*/
    int* h_data0;
    int* h_data1;
    int* h_result;
    h_data0 = (int*)calloc(size*3, sizeof(int));
    h_data1 = &h_data0[size];
    h_result = &h_data0[size*2];

    srand(990720);
    for (int i=0;i<size;i++){
        h_data0[i]= rand()%10;
    }
    // Aprint(h_data0, size);
    // Aprint(h_data1, size);

    /*init device memory*/
    int* d_data0;
    int* d_data1;
    int* d_result;
    CUDA_CHECK(hipMalloc((void**)&d_data0, (size*3)*sizeof(int)));
    d_data1=&d_data0[size];
    d_result=&d_data0[size*2];

    /*memcpyH2D*/
    CUDA_CHECK(hipMemcpyAsync(d_data0, h_data0, size*3*sizeof(int), hipMemcpyHostToDevice, stream0));

    hipGraphExec_t gExec1, gExec2, gExec3;
    hipGraphNode_t a, b,c;
    hipGraph_t g1, g2, g3;
    hipGraphCreate(&g2,0);

    CUDA_CHECK(hipStreamBeginCapture(stream0, hipStreamCaptureModeGlobal));
    adding(d_data0, stream0, size);
    CUDA_CHECK(hipStreamEndCapture(stream0, &g3));

    hipGraphAddChildGraphNode(&b, g2, NULL, 0, g3);
    struct hipKernelNodeParams kr={0};
    kr.func=(void*)relaunchSelf;
    kr.gridDim=1;
    kr.blockDim=1;
    kr.sharedMemBytes=0;
    kr.kernelParams=NULL;
    kr.extra=NULL;

    hipGraphAddKernelNode(&a, g2, &b,1,&kr);
    // hipGraphAddKernelNode(&c, g1, &a,2,&kr);

    hipGraphInstantiate(&gExec2, g2, hipGraphInstantiateFlagDeviceLaunch);
    hipGraphUpload(gExec2, stream0);

        //printf("%3d %lf\n",i, (double)((end - start)));
    hipStreamBeginCapture(stream0, hipStreamCaptureModeGlobal);
    launchTailGraph<<<1, 1, 0, stream0>>>(gExec2);
    hipStreamEndCapture(stream0, &g1);
    hipGraphInstantiate(&gExec1, g1);
    g_start = clock();
    hipGraphLaunch(gExec1, stream0);
    CUDA_CHECK(hipStreamSynchronize(stream0));
    g_end = clock();
    printf("total %lf s\n", ((float)(g_end - g_start)/1000000));
    
    /*memcpyD2H*/
    CUDA_CHECK(hipMemcpyAsync(h_data0, d_data0, size*3*sizeof(int), hipMemcpyDeviceToHost, stream0));
    Aprint(h_data0, size);
    // Aprint(h_data1, size);
    // Aprint(h_result, size);

    /*free*/
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    CUDA_CHECK(hipFree(d_data0));
    CUDA_CHECK(hipStreamDestroy(stream0));
    CUDA_CHECK(hipStreamDestroy(stream1));
    free(h_data0);
    return 0;
} 

