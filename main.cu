#include "hip/hip_runtime.h"
#include <func.hpp>

#define CUDA_CHECK(val) { \
	if (val != hipSuccess) { \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
		exit(val); \
	} \
}

using namespace std;

int main(){
    clock_t g_start, g_end;
    
    int size =1<<10;
    printf("%d\n",size);
    hipStream_t stream0;
    hipStream_t stream1;
    CUDA_CHECK(hipStreamCreate(&stream0));
    CUDA_CHECK(hipStreamCreate(&stream1));
    /*init host memory*/
    int* h_data0;
    int* h_data1;
    int* h_result;
    h_data0 = (int*)calloc(size*3, sizeof(int));
    h_data1 = &h_data0[size];
    h_result = &h_data0[size*2];

    srand(990720);
    for (int i=0;i<size;i++){
        h_data0[i]= rand()%10;
    }
    // Aprint(h_data0, size);
    // Aprint(h_data1, size);

    /*init device memory*/
    int* d_data0;
    int* d_data1;
    int* d_result;
    CUDA_CHECK(hipMalloc((void**)&d_data0, (size*3)*sizeof(int)));
    d_data1=&d_data0[size];
    d_result=&d_data0[size*2];

    /*memcpyH2D*/
    CUDA_CHECK(hipMemcpyAsync(d_data0, h_data0, size*3*sizeof(int), hipMemcpyHostToDevice, stream0));
    g_start = clock();
    for (int i=0; i<100000;i++){
    adding(d_data0, stream0, size);
    }
    g_end = clock();
    printf("total %lf s\n", ((float)(g_end - g_start)/1000000));

    /*memcpyD2H*/
    CUDA_CHECK(hipMemcpyAsync(h_data0, d_data0, size*3*sizeof(int), hipMemcpyDeviceToHost, stream0));
    Aprint(h_data0, size);
    // Aprint(h_data1, size);
    // Aprint(h_result, size);

    /*free*/
    CUDA_CHECK(hipFree(d_data0));
    CUDA_CHECK(hipStreamDestroy(stream0));
    CUDA_CHECK(hipStreamDestroy(stream1));
    free(h_data0);
    return 0;
} 

