#include "hip/hip_runtime.h"
#include <func.hpp>
std::vector<hipGraphNode_t> _node_list;
#define CUDA_CHECK(val) { \
	if (val != hipSuccess) { \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(val), __LINE__, __FILE__); \
		exit(val); \
	} \
}

using namespace std;


int main(){
    clock_t g_start, g_end;
    graph_created=false;
    int size =1<<10;
    printf("%d\n",size);
    hipStream_t stream0;
    hipStream_t stream1;
    CUDA_CHECK(hipStreamCreate(&stream0));
    CUDA_CHECK(hipStreamCreate(&stream1));
    /*init host memory*/
    int* h_data0;
    int* h_data1;
    int* h_result;
    h_data0 = (int*)calloc(size*3, sizeof(int));
    h_data1 = &h_data0[size];
    h_result = &h_data0[size*2];

    srand(990720);
    for (int i=0;i<size;i++){
        h_data0[i]= rand()%10;
    }
    // Aprint(h_data0, size);
    // Aprint(h_data1, size);

    /*init device memory*/
    int* d_data0;
    int* d_data1;
    int* d_result;
    CUDA_CHECK(hipMalloc((void**)&d_data0, (size*3)*sizeof(int)));
    d_data1=&d_data0[size];
    d_result=&d_data0[size*2];

    /*memcpyH2D*/
    CUDA_CHECK(hipMemcpyAsync(d_data0, h_data0, size*3*sizeof(int), hipMemcpyHostToDevice, stream0));

    hipGraphExec_t gExec1, gExec2, gExec3;
    hipGraphNode_t a, b,c;
    hipGraph_t g1, g2, g3;
    void *kernelArgs[1];

    hipGraphCreate(&g3,0);
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, g3, 1, cudaGraphCondAssignDefault);
    hipGraphNodeParams cParams = { hipGraphNodeTypeConditional };
    cParams.conditional.handle = handle;
    cParams.conditional.type   = cudaGraphCondTypeWhile;
    cParams.conditional.size   = 1;
    hipGraphAddNode(&a, g3, NULL, 0, &cParams);
    
    g2 = cParams.conditional.phGraph_out[0];

    hipGraphNodeParams params = { hipGraphNodeTypeKernel };
    params.kernel.func = (void *)loopKernel;
    params.kernel.gridDim.x = params.kernel.gridDim.y = params.kernel.gridDim.z = 1;
    params.kernel.blockDim.x = params.kernel.blockDim.y = params.kernel.blockDim.z = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &handle;
    
    CUDA_CHECK(hipStreamBeginCapture(stream0, hipStreamCaptureModeGlobal));
    adding(d_data0, stream0, size);
    CUDA_CHECK(hipStreamEndCapture(stream0, &g1));

    hipGraphAddNode(&c, g2, NULL, 0, &params);
    hipGraphAddChildGraphNode(&b, g2, &c, 1, g1);

    hipGraphInstantiate(&gExec1, g3, NULL, NULL, 0);
    g_start = clock();
    hipGraphLaunch(gExec1, 0);
    CUDA_CHECK(hipDeviceSynchronize());
    g_end = clock();
    printf("total %lf s\n", ((float)(g_end - g_start)/1000000));
    
    /*memcpyD2H*/
    CUDA_CHECK(hipMemcpyAsync(h_data0, d_data0, size*3*sizeof(int), hipMemcpyDeviceToHost, stream0));
    Aprint(h_data0, size);
    // Aprint(h_data1, size);
    // Aprint(h_result, size);

    /*free*/
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    CUDA_CHECK(hipFree(d_data0));
    CUDA_CHECK(hipStreamDestroy(stream0));
    CUDA_CHECK(hipStreamDestroy(stream1));
    free(h_data0);
    return 0;
} 

