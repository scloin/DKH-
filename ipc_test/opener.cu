//app 1, part of a 2-part IPC example                                                                                                                                                                                                                                                                                    

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#define DSIZE 28*28

#define cudaCheckErrors(msg) \
  do { \
  hipError_t __err = hipGetLastError(); \
  if (__err != hipSuccess) { \
  fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
    msg, hipGetErrorString(__err), \
          __FILE__, __LINE__); \
  fprintf(stderr, "*** FAILED - ABORTING\n"); \
  exit(1); \
  } \
  } while (0)

__global__ void polling(int* head){
  int check=1;
  while(check!=0){
    //wait for head==0
    check=atomicExch(&head[0],1);
  }
}

__global__ void set_tail(int* tail){
  atomicExch(&tail[0],0);
}

int main(){
  system("rm -f testfifo"); // remove any debris                                                                                                                                                                                                                                                                         
  int ret = mkfifo("testfifo", 0600); // create fifo                                                                                                                                                                                                                                                                     
  if (ret != 0) {printf("mkfifo error: %d\n",ret); return 1;}


  float h_nums[] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  float h_nums2[] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                    0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  float h_nums3[] = {-1};
  void *data;
  hipMalloc(&data, DSIZE*sizeof(float)+3*sizeof(int));
  int* out = (int *)((float*)data+DSIZE);
  int* head=(int *)((float*)data+DSIZE)+1;
  int* tail=(int *)((float*)data+DSIZE)+2;

  int h_out[]={9707, -1, -1};
  int hhead[]={9};
  hipIpcMemHandle_t my_handle;

  hipMemcpy(out, h_out, 3*sizeof(int), hipMemcpyHostToDevice);

  cudaCheckErrors("malloc fail");                                                                                                                                                                                                                                                        
  hipIpcGetMemHandle(&my_handle, data);
  unsigned char handle_buffer[sizeof(my_handle)+1];
  memset(handle_buffer, 0, sizeof(my_handle)+1);
  memcpy(handle_buffer, (unsigned char *)(&my_handle), sizeof(my_handle));
  cudaCheckErrors("get IPC handle fail");
  FILE *fp;
  printf("waiting for app2\n");
  fp = fopen("testfifo", "w");
  if (fp == NULL) {printf("fifo open fail \n"); return 1;}
  printf("%ld\n", sizeof(my_handle));
  for (int i=0; i < sizeof(my_handle); i++){
    ret = fprintf(fp,"%c", handle_buffer[i]);
    if (ret != 1) printf("ret = %d\n", ret);}
  fclose(fp);


  sleep(2); // wait for app 2 to modify data                                                                                                                                                                                                                                                                             
  //float *result = (float *)malloc(DSIZE*sizeof(float));
  
  // cudaMemcpy(result, data, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);
  printf("7\n");
  hipMemcpy((float*)data, h_nums, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  set_tail<<<1,1>>>(tail); 
  polling<<<1,1>>>(head);    
  hipDeviceSynchronize();
  printf("1\n");
  sleep(2);
  hipMemcpy((float*)data, h_nums2, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  set_tail<<<1,1>>>(tail);
  polling<<<1,1>>>(head); 
  hipDeviceSynchronize();
  printf("7\n");
  sleep(2);
  hipMemcpy((float*)data, h_nums, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  set_tail<<<1,1>>>(tail); 
  polling<<<1,1>>>(head);    
  hipDeviceSynchronize();
  printf("1\n");
  sleep(2);
  hipMemcpy((float*)data, h_nums2, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  set_tail<<<1,1>>>(tail);
  polling<<<1,1>>>(head); 
  hipDeviceSynchronize();
  printf("7\n");
  sleep(2);
  hipMemcpy((float*)data, h_nums, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  set_tail<<<1,1>>>(tail); 
  polling<<<1,1>>>(head);    
  hipDeviceSynchronize();
  printf("1\n");
  sleep(2);
  hipMemcpy((float*)data, h_nums2, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  set_tail<<<1,1>>>(tail);
  polling<<<1,1>>>(head); 
  hipDeviceSynchronize();
  printf("e\n");
  sleep(2);
  hipMemcpy((float*)data, h_nums3, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  //cudaMemcpy(result, (float*)data, DSIZE*sizeof(float), cudaMemcpyDeviceToHost);


  //printf("%lf\n", *result);
  // if (!(*result)) printf("Fail!\n");
  // else printf("Success!\n");
  system("rm testfifo");
  hipDeviceReset();
  return 0;
}